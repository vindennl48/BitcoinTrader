#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "../../mth.h"

typedef unsigned long int INT;


// block size and amount
#define T     1024
#define B(x)  (x+T-1)/T


// Kernel
////////////////////////////////////////////////////////////////////////////////
__global__
void add(INT *a, INT *b, INT *c, INT size){
  INT index = threadIdx.x + blockDim.x * blockIdx.x;
  if(index < size)
    c[index] = a[index] + b[index];
  //for(INT i=0; i<size; i+=1)
  //  c[i] = a[i] + b[i];
};
////////////////////////////////////////////////////////////////////////////////


int main(){
  INT N = /*elements*/ 2048*2048*10;
  cArray<INT> a(N), b(N), c(N);

  for(INT i=0; i<N; i+=1){
    a.h[i] = i;
    b.h[i] = i;
  }

  INT start = mtime();
  a.send(); b.send();
  add<<<B(N),T>>>(a.d, b.d, c.d, N);
  c.receive();
  INT end = mtime();

  print("\n" << "Elapsed Time: " << (start-end) << "ms" << "\n");


  for(INT i=(N-10); i<N; i+=1)
    std::cout << a.h[i] << ", ";
  std::cout << std::endl;

  for(INT i=(N-10); i<N; i+=1)
    std::cout << c.h[i] << ", ";

  return 0;
}
