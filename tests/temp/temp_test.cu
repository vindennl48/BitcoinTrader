#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <string>
#include "../../mth.h"

typedef unsigned long int UINT;


#undef  THREADS
#define THREADS  (1<<9)   /* 512 */

// KERNELS
////////////////////////////////////////////////////////////////////////////////
__global__
void kernel_prepare(double *sums, double *reacts,
    double *inputs, UINT ITR, UINT N){
  UINT tindex    = blockIdx.x*blockDim.x+threadIdx.x;
  if(tindex<N) sums[tindex] = inputs[ITR*N+tindex];
  reacts[tindex] = 1/(1+exp(-sums[tindex]))-.5;
  sums[tindex]   = 0;
};
__global__
void kernel_fire(double *sums, double *reacts, double *weights){
  volatile __shared__ double sdata[THREADS];
  volatile __shared__ double rdata[THREADS];
  volatile __shared__ double wdata[THREADS];

  UINT tid    = threadIdx.x;  /* opposing neuron */
  //UINT tindex = blockIdx.x*blockDim.x+threadIdx.x;
  UINT neuron = blockIdx.x;   /* this block's neuron */
  rdata[tid]  = reacts[tid];
  wdata[tid]  = weights[blockDim.x*tid+neuron];
  __syncthreads();

  if(tid != neuron)
    sdata[tid] = rdata[tid]*wdata[tid];
  else
    sdata[tid] = 0;
  __syncthreads();

  for(UINT s=blockDim.x/2; s>0; s>>=1){
    if(tid<s)
      sdata[tid]+=sdata[tid+s];
    __syncthreads();
  };

  if(tid==0) sums[blockIdx.x]=sdata[0];
};
__global__
void kernel_get_fit(double *sums, double *fitness, UINT num_neurons){
  UINT result = 1/(1+exp(-sums[num_neurons]))-.5;
  if(result > 0)
    /*buy*/
  else
    /*sell*/
};
////////////////////////////////////////////////////////////////////////////////


void get_candles(int, double*);


#define AGENT_RANDOM  2
struct Agent{
  UINT           agentID;
  UINT           num_neurons;
  double         trade[4];
  cArray<double> sums;
  cArray<double> reacts;
  cArray<double> weights;
  cArray<double> fitness;

  Agent(UINT num_neurons, UINT ID, UINT options=2)
  :sums(num_neurons),reacts(num_neurons),weights(pow(num_neurons,2)),
   num_neurons(num_neurons),fitness(1),agentID(ID)
  {
    if(options == AGENT_RANDOM){

      /* create a random brain */
      loop(i, num_neurons){
        sums.h[i] = 0;
        reacts.h[i] = 0;
      };
      loop(i, pow(num_neurons,2))
        weights.h[i] = ((double)(rand()%200)-100)/100;

      *fitness.h = 100;
    }
  };
  void send(){
    sums.send();
    reacts.send();
    weights.send();
    fitness.send();
  };
  void receive(){
    //sums.receive();
    fitness.receive();
  };
};


// MAIN
////////////////////////////////////////////////////////////////////////////////
int main(){

  const UINT num_neurons = THREADS;
  const UINT num_inputs  = 10000;
  cArray<double> candles(num_inputs);
  get_candles(num_inputs, candles.h);

  Agent agent(num_neurons, 2);

  int start = mtime();

  agent.send(); candles.send();
  loop(i, num_inputs){
    kernel_prepare<<<1,num_neurons>>>(
      agent.sums.d,
      agent.reacts.d,
      candles.d,
      i, 4
    );
    kernel_fire<<<num_neurons,num_neurons>>>(
      agent.sums.d,
      agent.reacts.d,
      agent.weights.d
    );
  };
  agent.receive();

  int end = mtime();

  loop(i, num_neurons)
    print("sum: " << agent.sums.h[i]);

  print("Time Elapsed: " << (start-end) << "ms, " << (double)(20*(start-end))/1000 << "sec/20");

  return 0;
}
////////////////////////////////////////////////////////////////////////////////

inline void
get_candles(int num_candles, double *candles){
  ifstream        if_candles;
  v(double)       candles_temp;
  //cArray<double>  candles(num_candles);

  if_candles.open("candles_raw.txt");
  string STRING;

  while(!if_candles.eof()){
    try{
      getline(if_candles, STRING);
      try{ candles_temp.push_back(atof(STRING.c_str()));} 
      catch(...){ print("Errir converting string to double."); };
      STRING = "";
    }
    catch(...){
      print("candles_raw.txt has an uneven amount of lines!");
    };
  };

  UINT size = candles_temp.size()-2;

  loop(i, num_candles){
    double t = candles_temp[size-i];
    candles[i] = t;
  };
};
