#include "hip/hip_runtime.h"
// first cuda program

#include <iostream>
#include <math.h>

__global__
void add(int N, double *x, double*y)
{
  for (int i = 0; i < N; i++)
    y[i] = x[i] + y[i];
}

int main()
{
  int N = 1<<20;
  double a = 5;
  double *x, *y;

  hipMallocManaged(&x, N*sizeof(double));
  hipMallocManaged(&y, N*sizeof(double));

  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0;
    y[i] = 2.0;
  }

  x[0] = &a;

  add<<<1, 1>>>(N, x, y);
  hipDeviceSynchronize();

  double maxError = 0.0;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);

  return 0;
}
