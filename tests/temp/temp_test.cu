#include "hip/hip_runtime.h"
// first cuda program

#include <iostream>
#include <math.h>

__global__
void add(int *a, int *b, int *c){
  *c = *a + *b;
}

int main(){
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = 2;
  b = 7;

  cudamemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  cudamemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  add<<<1,1>>>(d_a, d_b, d_c);

  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
