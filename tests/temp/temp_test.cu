// first cuda program


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int *a, int *b, int *c){
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#define N 512

int main(){
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);

  for(int i=0; i<N; i+=1){
    a[i] = i;
    b[i] = i;
  }

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  add<<<1,N>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  free(a);
  free(b);
  free(c);

  hipFree(d_a); hipFree(d_b); hipFree(d_c);


  return 0;
}
