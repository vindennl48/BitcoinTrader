#include "hip/hip_runtime.h"
#include "../../mth.h"

// GLOBAL VARS
//#define THREADS   (1<<7)  /* 7=128, 8=256, 9=512, 10=1024 */
//#define CANDLES   2000
#define Na  2        /*num agents*/
#define Nn  (1<<7)   /*num neurons per agent*/
#define Np  (Nn*Nn)  /*num processes per agent*/


struct Neuron{
  cArray<double> sum;
  cArray<double> react;
  cArray<double> weights;
  Neuron():sum(1),react(1),weights(Nn){
    sum[0]   = 1;
    react[0] = 0;
    loop(i, Nn)
      weights[i] = ((double)(rand()%200)-100)/100;
  };
};

struct Agent{
  cArray<Neuron> neurons;
  Agent():neurons(Nn){
    loop(i, Nn)
      neurons[i] = Neuron();
  };
};

struct SB{
  cArray< cArray<double> > neurons;
  SB():neurons(Nn){
    loop(i, Nn){
      neurons[i] = cArray<double>(Nn);
      loop(j, Nn)
        neurons[i][j] = 0;
    };
  };
}; /*sums block*/


#include "kernels.h"
// #include "environment.h"

int main(){

  cArray<SB> sb_all(Na);
  loop(i, Na)
    sb_all[i] = SB();

  cArray<Agent> agents(Na);
  loop(i, Na)
    agents[i] = Agent();

  // agents.send();
  // kernel_prepare<<<Na,Nn>>>(agents.d);
  // kernel_fire<<<(Nn*Na),Nn>>>(agents.d, sb_all.send());
  // sb_all.receive();

  // agents.receive();
  loop(i, Na){
    loop(j, 10)
      nprint(agents[i].neurons[j].sum[0] << ", ");
    print("\n");
    loop(j, 10)
      nprint(agents[i].neurons[j].react[0] << ", ");
    print("\n");
  };

  // loop(i, Na){
  //   loop(j, Np){
  //     nprint(sb_all[i].neurons[i][j] << ", ");
  //     PAUSE;
  //   };
  //   print("--------------------------------------------------------");
  // };


////////////////////////////////////////////////////////////////////////////////
  // Environment e(10);
  // int start = mtime();
  // loop(i, 100)
  //   e.run();
  // int end = mtime();
  // e.Print();

  // int time = (start-end);

  // print("Elapsed Time: " << time << "ms");
////////////////////////////////////////////////////////////////////////////////

  // const UINT num_neurons = THREADS;
  // const UINT num_candles = 10000;
  // UINT agentID = 0;
  // v(Agent) agents;

  // cArray<double> candles(num_candles*4);
  // get_candles(num_candles, &candles);
  // candles.send();

  // int start = mtime();
  // loop(i, 10){
  //   agents.push_back(Agent(agentID, num_neurons, &candles, num_candles));
  //   agentID+=1;
  //   agents.back().send();
  //   agents.back().get_fitness();
  //   agents.back().receive();
  //   print("Agent " << i << " fitness: " << agents.back().fitness[0]);
  // };
  // int end = mtime();

  // loop(i, 10){
  //   print("Agent " << i << " fitness: " << agents[i].fitness[0]);
  // };

  // print("Time Elapsed: " << (start-end) << "ms, " << (double)(100*(start-end))/1000/60 << "min/100gen");

  return 0;
}
