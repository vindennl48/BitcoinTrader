#include "hip/hip_runtime.h"


// GLOBAL VARS
#define THREADS   (1<<7)  /* 7=128, 8=256, 9=512, 10=1024 */
#define CANDLES   2000


#include "../../mth.h"
#include "environment.h"

int main(){

  Environment e(10);
  int start = mtime();
  loop(i, 100)
    e.run();
  int end = mtime();
  e.Print();

  int time = (start-end);

  print("Elapsed Time: " << time << "ms");

  // const UINT num_neurons = THREADS;
  // const UINT num_candles = 10000;
  // UINT agentID = 0;
  // v(Agent) agents;

  // cArray<double> candles(num_candles*4);
  // get_candles(num_candles, &candles);
  // candles.send();

  // int start = mtime();
  // loop(i, 10){
  //   agents.push_back(Agent(agentID, num_neurons, &candles, num_candles));
  //   agentID+=1;
  //   agents.back().send();
  //   agents.back().get_fitness();
  //   agents.back().receive();
  //   print("Agent " << i << " fitness: " << agents.back().fitness[0]);
  // };
  // int end = mtime();

  // loop(i, 10){
  //   print("Agent " << i << " fitness: " << agents[i].fitness[0]);
  // };

  // print("Time Elapsed: " << (start-end) << "ms, " << (double)(100*(start-end))/1000/60 << "min/100gen");

  return 0;
}
