#include "hip/hip_runtime.h"
#define USING_CUDA
#include "../../mth.h"

// GLOBAL VARS
#define Na  20       /*num agents*/
#define Nn  (1<<7)   /*128: num neurons per agent*/
#define Np  (Nn*Nn)  /*num processes per agent*/


cArray<double> sums(Na*Nn);
cArray<double> raw_sums(Na*Np);
cArray<double> reacts(Na*Nn);
cArray<double> weights(Na*Np);

__host__ inline void
send_agents(){
  sums.send(); raw_sums.send();
  reacts.send(); weights.send();
};

__host__ inline void
receive_agents(){
  sums.receive(); raw_sums.receive();
  reacts.receive(); weights.receive();
};

__device__ inline void
get_start_points(UINT *itr_rs, UINT *itr_w, const UINT &tidx){
  UINT a  = (UINT)tidx/Np; UINT n = tidx-a*Nn;
  *itr_rs = a*Nn+n;        *itr_w = a*Np+n;
};

__host__ inline void
set_random_agents(){
  loop(i, Na*Nn){
    sums[i]     = 1;
    raw_sums[i] = 0;
    reacts[i]   = 0;
  };
  loop(i, Na*Np)
    weights[i]=((double)(rand()%200)-100)/100;
};

#include "kernels.h"

int main(){
  set_random_agents();
  send_agents();

  loop(i, Nn) nprint(sums[i] << ", ");
  print("\n");

  kernel_prepare<<<Na,Nn>>>(sums.d, reacts.d);
  receive_agents();

  loop(i, Nn) nprint(sums[i] << ", ");
  print("\n");

  // kernel_fire<<<(Nn*Na),Nn>>>(agents.d, raw_sums.d);
  // raw_sums.receive();

  // loop(i, Na){
  //   loop(j, Np){
  //     nprint(sb_all[i].neurons[i][j] << ", ");
  //     PAUSE;
  //   };
  //   print("--------------------------------------------------------");
  // };


////////////////////////////////////////////////////////////////////////////////
  // Environment e(10);
  // int start = mtime();
  // loop(i, 100)
  //   e.run();
  // int end = mtime();
  // e.Print();

  // int time = (start-end);

  // print("Elapsed Time: " << time << "ms");
////////////////////////////////////////////////////////////////////////////////

  // const UINT num_neurons = THREADS;
  // const UINT num_candles = 10000;
  // UINT agentID = 0;
  // v(Agent) agents;

  // cArray<double> candles(num_candles*4);
  // get_candles(num_candles, &candles);
  // candles.send();

  // int start = mtime();
  // loop(i, 10){
  //   agents.push_back(Agent(agentID, num_neurons, &candles, num_candles));
  //   agentID+=1;
  //   agents.back().send();
  //   agents.back().get_fitness();
  //   agents.back().receive();
  //   print("Agent " << i << " fitness: " << agents.back().fitness[0]);
  // };
  // int end = mtime();

  // loop(i, 10){
  //   print("Agent " << i << " fitness: " << agents[i].fitness[0]);
  // };

  // print("Time Elapsed: " << (start-end) << "ms, " << (double)(100*(start-end))/1000/60 << "min/100gen");

  return 0;
}
